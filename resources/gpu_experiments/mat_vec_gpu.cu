#include <algorithm>
#include <assert.h>
#include <iostream>
#include <math.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <sys/time.h>
#include <hip/hip_runtime.h>

float* d_A = NULL;	// Pointer to matrix A in device memory
float* d_x = NULL;	// Pointer to vector x in device memory
float* d_y = NULL;	// Pointer to result vector y in device memory

double time_memcpy = 0;
double time_compute = 0;	// Timers

/////////////////////////////////////////////////////////////////////////
// Matrix-vector Multiplication on CPU: Policy 1 
/////////////////////////////////////////////////////////////////////////
void mv_cpu(float* y, const float* A, const float* x, int n) {
	for (int i=0; i<n; i++) {
		for (int j=0; j<n; j++) {
			y[i] += A[i*n + j] * x[j];
		}
	}	
}

/////////////////////////////////////////////////////////////////////////
// Matrix-vector Multiplication on GPU: Policy 2 
/////////////////////////////////////////////////////////////////////////
__global__ void mv_cuda_simple(float* y, float* A, float* x, int n)
{
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < n) {
		float temp = 0;		
		for (int k = 0; k < n; k++) {
			temp += A[row*n + k] * x[k];
		}
		y[row] = temp;
	}
}

/////////////////////////////////////////////////////////////////////////
// Matrix-vector Multiplication on GPU using shared memory: Policy 3 
/////////////////////////////////////////////////////////////////////////
__global__ void mv_cuda_shared(float* y, float* A, float* x, int n)
{
	extern __shared__ float s_x[];
	int row = blockIdx.x * blockDim.x + threadIdx.x;
	if (row < n) {
		float temp = 0;		
		for (int j = 0; j < n; j += blockDim.x) {
			if (j + threadIdx.x < n) {
				s_x[threadIdx.x] = x[j + threadIdx.x];
			}
			__syncthreads();

			for (int k = j; k < min(n, j + blockDim.x); k++) {
				temp += A[row*n + k] * s_x[k - j];
			}
			__syncthreads();
		}
		y[row] = temp;
	}
}

/////////////////////////////////////////////////////////////////////////
// Allocate device memory and copy from host to device memory
/////////////////////////////////////////////////////////////////////////
void copy_host_to_device(float* A, float* x, int n)
{
	double time_start, time_end;
	struct timeval tv;
	struct timezone tz;

	gettimeofday (&tv ,   &tz);
	time_start = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;

	hipMalloc((void**) &d_A, sizeof(float) * n * n);
	hipMalloc((void**) &d_x, sizeof(float) * n);
	hipMalloc((void**) &d_y, sizeof(float) * n);
	assert(d_A);
	assert(d_x);
	assert(d_y);
	hipMemcpy(d_A, A, sizeof(float) * n * n, hipMemcpyHostToDevice);
	hipMemcpy(d_x, x, sizeof(float) * n, hipMemcpyHostToDevice);

	gettimeofday (&tv ,   &tz);
	time_end = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;

	time_memcpy += time_end - time_start;
}

/////////////////////////////////////////////////////////////////////////
// Copy result from device to host memory and free device memory
/////////////////////////////////////////////////////////////////////////
void copy_device_to_host(float* y, int n)
{
	double time_start, time_end;
	struct timeval tv;
	struct timezone tz;

	gettimeofday (&tv ,   &tz);
	time_start = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;

	hipMemcpy(y, d_y, sizeof(float) * n, hipMemcpyDeviceToHost);
	hipFree(d_A);
	hipFree(d_x);
	hipFree(d_y);

	gettimeofday (&tv ,   &tz);
	time_end = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;

	time_memcpy += time_end - time_start;
}

/////////////////////////////////////////////////////////////////////////
// Program main
/////////////////////////////////////////////////////////////////////////
int main(int argc, char** argv)
{
	double time_start, time_end;
	struct timeval tv;
	struct timezone tz;

	if(argc != 5) {printf("Usage: mat_vec_gpu <array size> <block size> <policy (1|2|3)> <(debug|nodebug)>\n") ; exit(2) ;}
	int N = atoi(argv[1]) ;				// array size
	int block_size = atoi(argv[2]) ;	// size of thread block
	int policy = atoi(argv[3]) ;		// 1, 2 or 3
	bool debug = (!strcmp(argv[4], "debug")) ? true : false;

	if(debug) {
		printf("\n Array size = %d , block size = %d and policy = %d\n ", N, block_size, policy);
	}

	// set seed for rand()
	srand(2020);

	// allocate host memory for matrix A and vector x
	int matrix_size = N * N;
	float* h_A = (float*)malloc(sizeof(float) * matrix_size);
	float* h_x = (float*)malloc(sizeof(float) * N);

	// randomly initialize host memory
	for (int i = 0; i < matrix_size; ++i){
		h_A[i] = rand() / (float)RAND_MAX;
	}
	for (int i = 0; i < N; i++){
		h_x[i] = rand() / (float)RAND_MAX;
	}

	// allocate host memory for the result
	float* h_y = (float*)malloc(sizeof(float) * N);
	for (int i = 0; i < N; i++) {
		h_y[i] = 0.0;
	}

	//call the relevant policy 
	if(policy==1){ //CPU baseline
		gettimeofday (&tv ,   &tz);
		time_start = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;

		mv_cpu(h_y, h_A, h_x, N);

		gettimeofday (&tv ,   &tz);
		time_end = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;
		time_compute = time_end - time_start;
	}
	else if(policy==2){ //CUDA MV multiplication
		// call mv_cuda_simple <<< >>> ( )

		copy_host_to_device(h_A, h_x, N);

		gettimeofday (&tv ,   &tz);
		time_start = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;

		int nblocks = (N + block_size - 1) / block_size;
		mv_cuda_simple <<<nblocks, block_size>>> (d_y, d_A, d_x, N);
		hipDeviceSynchronize();

		gettimeofday (&tv ,   &tz);
		time_end = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;
		time_compute = time_end - time_start;

		copy_device_to_host(h_y, N);
	}
	else if(policy==3){ //CUDA MV multiplication with shared memory
		// call mv_cuda_shared<<<grid, block_size >>>(d_y, d_A, d_x, N);

		copy_host_to_device(h_A, h_x, N);

		gettimeofday (&tv ,   &tz);
		time_start = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;

		int nblocks = (N + block_size - 1) / block_size;
		mv_cuda_shared <<<nblocks, block_size, sizeof(float) * block_size>>> (d_y, d_A, d_x, N);
		hipDeviceSynchronize();

		gettimeofday (&tv ,   &tz);
		time_end = (double)tv.tv_sec + (double)tv.tv_usec / 1000000.0;
		time_compute = time_end - time_start;

		copy_device_to_host(h_y, N);
	}

	if (debug) {
		// print selected results (for checking correctness) and print time elapsed
		for (int i=0; i<N; i+=N/10) { printf(" %10.6f",h_y[i]); }
		printf("\n");
	}
	printf("Memcpy Time: %f seconds\n", time_memcpy);
	printf("Compute Time: %f seconds\n", time_compute);

	// clean up memory allocated  
	free(h_A);
	free(h_x);
	free(h_y);

	hipDeviceReset();
}
